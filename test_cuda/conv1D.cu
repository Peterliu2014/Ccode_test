#include<cstdio>
#include<hip/hip_runtime.h>
#include<vector>
#include<string>

#define BLOCK_SIZE 1024

using namespace std;

float average(const vector<float> &timing) {
  double avg = 0;
  for(vector<float>::const_iterator it = timing.begin(); it != timing.end(); it++) avg += *it;
  avg /= timing.size();

  //return us
  avg /= 1000;

  return avg;
}

void print_info(float *data, int len, string flag) {
    printf("%s frist ten:\n", flag.c_str());
    for (int i=0; i<10; i++){
        printf("%.4lf ", data[i]);
    }
    printf("\n");
    printf("%s last ten:\n", flag.c_str());
    for (int i=len -10; i<len; i++){
        printf("%.4lf ", data[i]);
    }
    printf("\n");
}

__global__ void convDirect(float *A, float*B, float*C, int len_a, int len_b, int len_c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid > len_c) return;
    float sum =0.0f;
    for (int i=0; i<len_b; i++) {
        int idx = tid - i;
        if (idx >=0 && idx < len_a) {
            sum += A[idx] * B[i];
        }
    }
    C[tid] = sum;
}

__global__ void conv_shared_memory(float *A, float *B, float *C, int len_a, int len_b, int len_c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid > len_c) return;
    int tx = threadIdx.x;
    extern __shared__ float sA[];
    if (tid < len_a) {
        sA[tx] = A[tid];
    }
    __syncthreads();
    float sum = 0.0f;
    for (int i=0; i<len_b; i++) {
        int idx = tid - i;
        int sx = tx - i;
        if (tid < len_a && sx>=0) {
            sum += sA[sx] * B[i];
        }
        else if (idx >=0 && idx < len_a) {
            sum += A[idx] * B[i];
        }
    }
    __syncthreads();
    C[tid] = sum;
}

__constant__ static float c_B[1024];
__global__ void conv_shared_const_memory(float *A, float*B, float *C, int len_a, int len_b, int len_c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;
    if (tid > len_c) return;
    extern __shared__ float sA[];
    if (tid < len_a) {
        sA[tx] = A[tid];
    }
    __syncthreads();
    float sum = 0.0f;
    for (int i=0; i<len_b; i++) {
        int idx = tid - i;
        int sx = tx - i;
        if (tid < len_a && sx >= 0) {
            sum += sA[sx] * c_B[i];
        }
        else if (idx >= 0 && idx < len_a) {
            sum += A[idx] *c_B[i];
        }
    }
    __syncthreads();
    C[tid] = sum;
}

int main() {
    int len_a = 1024 * 1024 * 256;
    int len_b = 1000;
    int len_c = len_a + len_b - 1;
    int numThreads = BLOCK_SIZE;
    int numBlocks = (len_c + numThreads - 1) / numThreads;

    float* A = (float*) malloc(len_a * sizeof(float));
    float* B = (float*) malloc(len_b * sizeof(float));
    float* C = (float*) malloc(len_c * sizeof(float));
    // file a,b
    for (int i=0; i<len_a; i++) A[i] = i;
    for (int i=0; i<len_b; i++) B[i] = i;
    // print_info(A, "A");
    // print_info(B, "B");

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, len_a * sizeof(float));
    hipMalloc((void**)&d_B, len_b * sizeof(float));
    hipMalloc((void**)&d_C, len_c * sizeof(float));

    hipMemcpy(d_A, A, len_a * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, len_b * sizeof(float), hipMemcpyHostToDevice);

    vector<float> times;
    int loops = 3;
    for (int i=0; i<loops; i++) {
        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start, NULL);

        convDirect<<<numBlocks, numThreads>>>(d_A, d_B, d_C, len_a, len_b, len_c);

        // conv_shared_memory<<<numBlocks, numThreads, numThreads * sizeof(float)>>>(d_A, d_B, d_C, len_a, len_b, len_c);

        // cudaMemcpyToSymbol(c_B, d_B, len_b * sizeof(float));
        // conv_shared_memory<<<numBlocks, numThreads, numThreads * sizeof(float)>>>(d_A, d_B, d_C, len_a, len_b, len_c);

        hipEventRecord(end, NULL);
        hipEventSynchronize(end);
        float time = 0;
        hipEventElapsedTime(&time, start, end);
        times.push_back(time);
    }
    printf("convDirect avg time:%lf\n", average(times));
    
    hipMemcpy(C, d_C, len_c * sizeof(float), hipMemcpyDeviceToHost);
    print_info(C, len_c, "C");

}