#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>

// unroll the last warp
__device__ void warpFunc(volatile int* sA, int tid) {
    sA[tid] += sA[tid+32];
    sA[tid] += sA[tid+16];
    sA[tid] += sA[tid+8];
    sA[tid] += sA[tid+4];
    sA[tid] += sA[tid+2];
    sA[tid] += sA[tid+1];
}

__global__ void reduce_kernel3(int*A, int*out, int len_a) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ volatile int sA[];
    sA[tid] = 0;
    if (idx < len_a) sA[tid] = A[idx];
    __syncthreads();
    for (int s=blockDim.x/2; s>32; s>>=1) {
        if(tid <s) {
          sA[tid] += sA[tid+s];
        }
        __syncthreads();
    }
    if (tid<32) warpFunc(sA, tid);
    if (tid==0) out[bid] = sA[0];
}

__global__ void reduce_kernel3_2(int*A, int*out, int len_a) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ volatile int sA[];
    sA[tid] = 0;
    if (idx < len_a) sA[tid] = A[idx];
    __syncthreads();
    for (int s=blockDim.x/2; s>32; s>>=1) {
        if(tid <s) {
          sA[tid] += sA[tid+s];
        }
        __syncthreads();
    }
    // only sA is volatile can do this
    if (tid<32) {
        sA[tid] += sA[tid+32];
        sA[tid] += sA[tid+16];
        sA[tid] += sA[tid+8];
        sA[tid] += sA[tid+4];
        sA[tid] += sA[tid+2];
        sA[tid] += sA[tid+1];
    }
    if (tid==0) out[bid] = sA[0];
}


int sum_on_cpu(int *data, int len) {
    int result = 0;
    for (int i=0; i<len; i++) result += data[i];
    return result;
}

int sum_on_gpu(int *data, int len, int numThreads, int flag) {
    int numBlocks = (len + numThreads -1) / numThreads;
    int*tmp;
    hipMalloc((void**)&tmp, numBlocks*sizeof(int));
    while(len>1) {
        if (flag==0) {
            reduce_kernel3<<<numBlocks, numThreads, numThreads*sizeof(int)>>>(data, tmp, len);
        }
        else if (flag==1){
            reduce_kernel3_2<<<numBlocks, numThreads, numThreads*sizeof(int)>>>(data, tmp, len);
        }
        else{
            reduce_kernel3_3<<<numBlocks, numThreads, numThreads*sizeof(int)>>>(data, tmp, len);
        }
        
        len = numBlocks;
        numBlocks = (numBlocks+numThreads-1)/numThreads;
        hipMemcpy(data, tmp, sizeof(int) * len, hipMemcpyDeviceToDevice);
    }
    int result;
    hipMemcpy(&result, tmp, sizeof(int), hipMemcpyDeviceToHost);
    return result;
}

int main() {
    int len = 1024 * 1024;
    int numThreads = 256;
    int *h_data = (int*) malloc(len * sizeof(int));
    std::fill_n(h_data, len, 1);

    int* d_data1, *d_data2, *d_data3;
    hipMalloc((void**)&d_data1, len * sizeof(int));
    hipMemcpy(d_data1, h_data, len*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_data2, len * sizeof(int));
    hipMemcpy(d_data2, h_data, len*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_data3, len * sizeof(int));
    hipMemcpy(d_data3, h_data, len*sizeof(int), hipMemcpyHostToDevice);
    
    int cpu_result = sum_on_cpu(h_data, len);
    int gpu_result1 = sum_on_gpu(d_data1, len, numThreads, 0);
    int gpu_result2 = sum_on_gpu(d_data2, len, numThreads, 1);
    int gpu_result3 = sum_on_gpu(d_data3, len, numThreads, 2);
    printf("result: cpu:%d, gpu k1:%d, k2:%d, k3:%d\n", cpu_result, gpu_result1, gpu_result2, gpu_result3);

    hipFree(d_data1);
    hipFree(d_data2);
    hipFree(d_data3);
    free(h_data);

}