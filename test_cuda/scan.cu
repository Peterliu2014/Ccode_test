#include "hip/hip_runtime.h"
#include<cstdio>
#include<vector>
#include<string>
#include<hip/hip_runtime.h>
// #include <thrust/sequence.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h> // thrust::host/device

#define BLOCK_SIZE 32

using namespace std;

float average(const vector<float> &timing) {
  double avg = 0;
  for(vector<float>::const_iterator it = timing.begin(); it != timing.end(); it++) avg += *it;
  avg /= timing.size();

  //return us
  avg /= 1000;

  return avg;
}

void print_info(int *data, int len, string flag) {
    printf("%s frist ten:\n", flag.c_str());
    for (int i=0; i<10; i++){
        printf("%d ", data[i]);
    }
    printf("\n");
    printf("%s last ten:\n", flag.c_str());
    for (int i=len -10; i<len; i++){
        printf("%d ", data[i]);
    }
    printf("\n");
}

__global__ void exclusive_scan(int *A, int N) {
    int thx = blockDim.x * blockIdx.x + threadIdx.x;
    if (thx >= N) return;
    __shared__ int logN;
    if (thx == 0) {
        logN = log2f(N);
    }
    __syncthreads();
    for (int i=0; i<logN; i++) {
        int d2 = powf(2, i+1);
        int dd2 = d2/2;
        if (thx%d2==d2-1) {
            A[thx] += A[thx - dd2];
        }
        __syncthreads();
    }
    if (thx == 0) {
        A[N-1] = 0;
    }
    __syncthreads();
    for (int i=logN-1; i>=0; i--) {
        int d2 = powf(2, i+1);
        int dd2 = d2/2;
        if (thx%d2==d2-1) {
            int tmp = A[thx-dd2];
            // A[thx-dd2] = tmp;
            A[thx-dd2] = A[thx];
            A[thx] += tmp;
        }
        __syncthreads();
    }
}

int main() {
    int len_a = BLOCK_SIZE;
    int numThreads = BLOCK_SIZE;
    int numBlocks = (len_a + numThreads - 1) / numThreads;

    int* A = (int*) malloc(len_a * sizeof(int));
    int* th_h_A = (int*) malloc(len_a * sizeof(int));
    // file a,b
    for (int i=0; i<len_a; i++) A[i] = i;
    for (int i=0; i<len_a; i++) th_h_A[i] = i;
    print_info(A, len_a, "A");

    int *d_A, *th_A;
    hipMalloc((void**)&d_A, len_a * sizeof(int));
    hipMalloc((void**)&th_A, len_a * sizeof(int));

    hipMemcpy(d_A, A, len_a * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(th_A, A, len_a * sizeof(int), hipMemcpyHostToDevice);

    // thrust scan
    thrust::exclusive_scan(thrust::device, th_A, th_A+len_a, th_A, 0);
    hipMemcpy(th_h_A, th_A, len_a * sizeof(int), hipMemcpyDeviceToHost);
    // thrust::exclusive_scan(thrust::host, th_h_A, th_h_A+len_a, th_h_A, 0);

    print_info(th_h_A, len_a, "th_h_A");

    vector<float> times;
    int loops = 1;
    for (int i=0; i<loops; i++) {
        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start, NULL);

        exclusive_scan<<<numBlocks, numThreads>>>(d_A, len_a);

        hipEventRecord(end, NULL);
        hipEventSynchronize(end);
        float time = 0;
        hipEventElapsedTime(&time, start, end);
        times.push_back(time);
    }
    printf("exclusive scan avg time:%lf\n", average(times));
    
    hipMemcpy(A, d_A, len_a * sizeof(float), hipMemcpyDeviceToHost);
    print_info(A, len_a, "A");

}