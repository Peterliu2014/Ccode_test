#include "hip/hip_runtime.h"
#include<cstdio>
#include<vector>
#include<string>
#include<hip/hip_runtime.h>
#include <thrust/sequence.h>
#include <thrust/scan.h>

#define BLOCK_SIZE 32

using namespace std;

float average(const vector<float> &timing) {
  double avg = 0;
  for(vector<float>::const_iterator it = timing.begin(); it != timing.end(); it++) avg += *it;
  avg /= timing.size();

  //return us
  avg /= 1000;

  return avg;
}

void print_info(float *data, int len, string flag) {
    printf("%s frist ten:\n", flag.c_str());
    for (int i=0; i<10; i++){
        printf("%.4lf ", data[i]);
    }
    printf("\n");
    printf("%s last ten:\n", flag.c_str());
    for (int i=len -10; i<len; i++){
        printf("%.4lf ", data[i]);
    }
    printf("\n");
}

__global__ void exclusive_scan(int *A, int N) {
    int thx = threadIdx.x;
    if (thx >= N) return;
    int logN = 0;
    if (thx == 0) {
        logN = log2f(N);
    }
    __syncthreads();
    for (int i=0; i<logN; i++) {
        int d2 = powf(2, i+1);
        int dd2 = d2/2;
        if (thx%d2==d2-1) {
            A[thx] += A[thx - dd2];
        }
        __syncthreads();
    }
    if (thx == 0) {
        A[N-1] = 0;
    }
    __syncthreads();
    for (int i=logN-1; i>=0; i--) {
        int d2 = powf(2, i+1);
        int dd2 = d2/2;
        if (thx%d2==d2-1) {
            int tmp = A[thx-dd2];
            A[thx-dd2] = tmp;
            A[thx] += tmp;
        }
        __syncthreads();
    }
}

int main() {
    int len_a = BLOCK_SIZE;
    int numThreads = BLOCK_SIZE;
    int numBlocks = (len_a + numThreads - 1) / numThreads;

    float* A = (float*) malloc(len_a * sizeof(float));
    float* th_h_A = (float*) malloc(len_a * sizeof(float));
    // file a,b
    for (int i=0; i<len_a; i++) A[i] = i;
    for (int i=0; i<len_a; i++) th_h_A[i] = i;
    print_info(A, "A");

    float *d_A, *th_A;
    hipMalloc((void**)&d_A, len_a * sizeof(float));
    hipMalloc((void**)&th_A, len_a * sizeof(float));

    hipMemcpy(d_A, A, len_a * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(th_A, A, len_a * sizeof(float), hipMemcpyHostToDevice);

    // thrust scan
    thrust::exclusive_scan(th_A, th_A+len_a, th_A);
    hipMemcpy(th_h_A, th_A, len_a * sizeof(float), hipMemcpyDeviceToHost);

    print_info(th_h_A, len_a, "th_h_A");

    vector<float> times;
    int loops = 3;
    for (int i=0; i<loops; i++) {
        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start, NULL);

        exclusive_scan<<<numBlocks, numThreads>>>(d_A, len_a);

        hipEventRecord(end, NULL);
        hipEventSynchronize(end);
        float time = 0;
        hipEventElapsedTime(&time, start, end);
        times.push_back(time);
    }
    printf("exclusive scan avg time:%lf\n", average(times));
    
    hipMemcpy(A, d_A, len_a * sizeof(float), hipMemcpyDeviceToHost);
    print_info(A, len_a, "A");

}