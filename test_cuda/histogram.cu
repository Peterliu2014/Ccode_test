#include "hip/hip_runtime.h"
#include<cstdio>
#include<vector>
#include<string>
#include<hip/hip_runtime.h>
#include<thrust/fill.h>


#define BLOCK_SIZE 256

using namespace std;

float average(const vector<float> &timing) {
  double avg = 0;
  for(vector<float>::const_iterator it = timing.begin(); it != timing.end(); it++) avg += *it;
  avg /= timing.size();

  return avg;
}


void print_info(int *data, int len, string flag) {
    printf("%s frist ten:\n", flag.c_str());
    for (int i=0; i<10; i++){
        printf("%d ", data[i]);
    }
    printf("\n");
    printf("%s last ten:\n", flag.c_str());
    for (int i=len -10; i<len; i++){
        printf("%d ", data[i]);
    }
    printf("\n");
}

void assign_value(int *data, int len) {
    for (int i=0; i<len; i++) data[i] = i % 256;
}

__global__ void naive_kernel(int *data, int *out, int len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    atomicAdd(&(out[data[tid]]), 1);
}

__global__ void histogram_kernel(int *data, int *out, int len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int thx = threadIdx.x;
    extern __shared__ int sdata[];
    sdata[thx] = 0;
    __syncthreads();

    if (tid<len) atomicAdd(&(sdata[data[tid]]), 1);
    __syncthreads();

    atomicAdd(&(out[thx]), sdata[thx]);
}

__global__ void histogram_kernel_multiple_read(int *data, int *out, int len, int stride) {
    int tid = blockIdx.x * (blockDim.x * stride) + threadIdx.x;
    int thx = threadIdx.x;
    extern __shared__ int sdata[];
    sdata[thx] = 0;
    __syncthreads();
    int end = min(tid + blockDim.x * stride, len);
    for (int i=tid; i< end; i+=blockDim.x) {
        atomicAdd(&(sdata[data[i]]), 1);
    }
    __syncthreads();

    atomicAdd(&(out[thx]), sdata[thx]);
}

void cal_histogram(int *data, int *out, int len, int numThreads) {
    int stride = 32;
    int numblocks = (len + (numThreads*stride) -1) / (numThreads * stride);
    // naive_kernel<<<numblocks, numThreads>>>(data, out, len);
    // histogram_kernel<<<numblocks, numThreads, 256 * sizeof(int)>>>(data, out, len);
    histogram_kernel_multiple_read<<<numblocks, numThreads, 256 * sizeof(int)>>>(data, out, len, stride);
}

void cal_histogram_cpu(int *data, int *out, int len) {
    for (int i=0; i<len; i++) out[data[i]]++;
}

int main() {
    int numThreads = BLOCK_SIZE;
    int len = 512 * 1024 * 1024;
    int *h_data = (int*)malloc(len * sizeof(int));
    assign_value(h_data, len);

    int *d_data;
    hipMalloc((void**)&d_data, len * sizeof(int));
    hipMemcpy(d_data, h_data, len * sizeof(int), hipMemcpyHostToDevice);

    int *result = (int*) malloc(256 * sizeof(int));
    fill_n(result, 256, 0);
    cal_histogram_cpu(h_data, result, len);
    print_info(result, 256, "cpu result");

    int *d_result;
    hipMalloc((void**)&d_result, 256 * sizeof(int));
    // thrust::fill(d_result, d_result + 256, 0);
    fill_n(result, 256, 0);
    hipMemcpy(d_result, result, 256 * sizeof(int), hipMemcpyHostToDevice);

    vector<float> times;
    int loops = 1;
    for (int i=0; i<loops; i++) {
        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start, NULL);

        cal_histogram(d_data, d_result, len, numThreads);
        
        hipEventRecord(end, NULL);
        hipEventSynchronize(end);
        float time = 0;
        hipEventElapsedTime(&time, start, end);
        times.push_back(time);
    }
    printf("cal histogram avg time:%lf\n", average(times));

    hipMemcpy(result, d_result, 256 * sizeof(int), hipMemcpyDeviceToHost);
    print_info(result, 256, "gpu result");

    hipFree(d_data);
    hipFree(d_result);
    free(h_data);
    free(result);
}