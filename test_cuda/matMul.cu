#include <cstdio>
#include <string>
#include<hip/hip_runtime.h>
#include<sys/time.h>
#include<vector>

using namespace std;

void print_data(float* arr, int width, int length, string flag) {
    int count=8;
    printf("%s :\n", flag.c_str());
    for (int i=0; i<count; i++) {
        for(int j=0; j<count; j++) {
            printf("%.3lf ", arr[i*length +i]);
        }
        printf("\n");
    }
}

float average(const vector<float> &timing) {
  double avg = 0;
  for(vector<float>::const_iterator it = timing.begin(); it != timing.end(); it++) avg += *it;
  avg /= timing.size();

  return avg;
}

void fillMat(float *mat, size_t rows, size_t cols)
{
    for(int row = 0; row < rows; ++row)
    {
        for(int col = 0; col < cols; ++col)
        {
            mat[row * cols + col] = col;
        }
    }
}


__global__ void matrixMulCUDA(float *C, float *A, float*B, int wA, int wB) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    float tmp = 0.0f;
    for (int i=0; i<wA; i++) {
        tmp += A[y * wA + i] * A[i * wB + x];
    }
    C[y * wB + x] = tmp;
}

template <int BLOCK_SIZE>
__global__ void matrixMulCUDA_share(float *C, float *A, float *B, int wA, int wB) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = by * wA;
    int aEnd = aBegin + wA -1;
    int aStep = BLOCK_SIZE;
    int bBegin = bx * BLOCK_SIZE;
    int bStep = wB * BLOCK_SIZE;

    float cSub = 0;
    __shared__ float subA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float subB[BLOCK_SIZE][BLOCK_SIZE];
    for (int a = aBegin, b = bBegin; a < aEnd; a+=aStep, b+=bStep) {
        subA[ty][tx] = A[a + ty * wA + tx];
        subB[ty][tx] = B[b + ty * wB + tx];
        __syncthreads();
        for (int i=0; i<BLOCK_SIZE; i++) {
            cSub += subA[ty][i] * subB[i][tx];
        }
        __syncthreads();
    }
    C[((by*BLOCK_SIZE+ty)*wB + bx*BLOCK_SIZE+tx)] = cSub;
}

template<int WIDTH>
void testMatrixMul()
{
    size_t mem_size = sizeof(float) * WIDTH * WIDTH;
    float *mat1 = (float *)malloc(mem_size);
    float *mat2 = (float *)malloc(mem_size);
    float *mat3 = (float *)malloc(mem_size);
    // initialize mat1 and mat2
    fillMat(mat1, WIDTH, WIDTH);
    fillMat(mat2, WIDTH, WIDTH);

    // matrixMul<WIDTH>(mat3, mat1, mat2);

    float* d_mat1, *d_mat2, *d_mat3;
    hipError_t error;
    error = hipMalloc((void**)&d_mat1, mem_size);
    error = hipMemcpy(d_mat1, mat1, mem_size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("hipMemcpy d_mat1 returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(1);
    }
    error = hipMalloc((void**)&d_mat2, mem_size);
    error = hipMemcpy(d_mat2, mat2, mem_size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("hipMemcpy d_mat2 returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(1);
    }
    error = hipMalloc((void**)&d_mat3, mem_size);

    int block_size = 32;
    dim3 threads(block_size, block_size);
    dim3 grid(WIDTH/threads.x, WIDTH/threads.y);
    // Allocate CUDA events that we'll use for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, NULL);

    vector<float> times;
    int loops = 3;
    for (int i=0;i<loops; i++) {
        hipEventRecord(start, NULL);
        matrixMulCUDA<<<grid, threads>>>(d_mat3, d_mat1, d_mat2, WIDTH, WIDTH);
        // if (block_size ==16) {
        //     matrixMulCUDA_share<16><<<grid, threads>>>(d_mat3, d_mat1, d_mat2, WIDTH, WIDTH);
        // }
        // else{
        //     matrixMulCUDA_share<32><<<grid, threads>>>(d_mat3, d_mat1, d_mat2, WIDTH, WIDTH);
        // }
        
        hipEventRecord(stop, NULL);
        hipEventSynchronize(stop);
        float time = 0;
        hipEventElapsedTime(&time, start, stop);
        times.push_back(time);
    }
    float avg_time = average(times);
    printf("cuda %d time: %lf ms\n", WIDTH, avg_time);
    

    // cudaEventRecord(stop, NULL);
    // cudaEventSynchronize(stop);
    // float time = 0;
    // cudaEventElapsedTime(&time, start, stop);
    // printf("run time:%.lf ms\n", time/loops);

    error = hipMemcpy(mat3, d_mat3, mem_size, hipMemcpyDeviceToHost);

    print_data(mat1, WIDTH, WIDTH, "mat1");
    print_data(mat2, WIDTH, WIDTH, "mat2");
    print_data(mat3, WIDTH, WIDTH, "mat3");

    free(mat1);
    free(mat2);
    free(mat3);
    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_mat3);
}

int main(int argc, char const *argv[])
{
    // testMatrixMul<256 >();
    // testMatrixMul<512 >();
    // testMatrixMul<768 >();
    testMatrixMul<2048>();
    // testMatrixMul<1280>();
    // testMatrixMul<1536>();
    // testMatrixMul<1792>();
    // testMatrixMul<2048>();
    return 0;
}
