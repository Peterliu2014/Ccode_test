#include "hip/hip_runtime.h"
#include <iostream>
#include<hip/hip_runtime.h>

using namespace std;

void fillMat(float *mat, size_t rows, size_t cols)
{
    for(int row = 0; row < rows; ++row)
    {
        for(int col = 0; col < cols; ++col)
        {
            mat[row * cols + col] = col;
        }
    }
}


__global__ void matrixMulCUDA(float *C, float *A, float*B, int wA, int wB) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    float tmp = 0.0f;
    for (int i=0; i<wA; i++) {
        tmp += A[x * wA + i] * A[i * wB + y];
    }
    C[x * wB + y] = tmp;
}

template<int WIDTH>
void testMatrixMul()
{
    size_t mem_size = sizeof(float) * WIDTH * WIDTH;
    float *mat1 = (float *)malloc(mem_size);
    float *mat2 = (float *)malloc(mem_size);
    float *mat3 = (float *)malloc(mem_size);
    // initialize mat1 and mat2
    fillMat(mat1, WIDTH, WIDTH);
    fillMat(mat2, WIDTH, WIDTH);

    // matrixMul<WIDTH>(mat3, mat1, mat2);

    float* d_mat1, *d_mat2, *d_mat3;
    hipError_t error;
    error = hipMalloc((void**)d_mat1, mem_size);
    error = codaMemcpy(d_mat1, mat1, mem_size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("hipMemcpy d_mat1 returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(1);
    }
    error = hipMalloc((void**)d_mat2, mem_size);
    error = codaMemcpy(d_mat2, mat2, mem_size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("hipMemcpy d_mat2 returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(1);
    }
    error = hipMalloc((void**)d_mat3, mem_size);
    error = codaMemcpy(d_mat3, mat3, mem_size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("hipMemcpy d_mat3 returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(1);
    }

    int block_size = 32;
    dim3 threads(block_size, block_size);
    dim3 grid(WIDTH/threads.x, WIDTH/threads.y);
    matrixMulCUDA<<<grid, threads>>>(d_mat3, d_mat1, d_mat2, WIDTH, WIDTH);

    free(mat1);
    free(mat2);
    free(mat3);
}

int main(int argc, char const *argv[])
{
    // testMatrixMul<256 >();
    // testMatrixMul<512 >();
    // testMatrixMul<768 >();
    testMatrixMul<1024>();
    // testMatrixMul<1280>();
    // testMatrixMul<1536>();
    // testMatrixMul<1792>();
    // testMatrixMul<2048>();
    return 0;
}