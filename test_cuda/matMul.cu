#include <cstdio>
#include <string>
#include<hip/hip_runtime.h>

using namespace std;

void print_data(float* arr, int width, int length, string flag) {
    int count=8;
    printf("%s :\n", flag.c_str());
    for (int i=0; i<count; i++) {
        for(int j=0; j<count; j++) {
            printf("%.3lf ", arr[i*length +i]);
        }
        printf("\n");
    }
}

void fillMat(float *mat, size_t rows, size_t cols)
{
    for(int row = 0; row < rows; ++row)
    {
        for(int col = 0; col < cols; ++col)
        {
            mat[row * cols + col] = col;
        }
    }
}


__global__ void matrixMulCUDA(float *C, float *A, float*B, int wA, int wB) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    float tmp = 0.0f;
    for (int i=0; i<wA; i++) {
        tmp += A[x * wA + i] * A[i * wB + y];
    }
    C[x * wB + y] = tmp;
}

template<int WIDTH>
void testMatrixMul()
{
    size_t mem_size = sizeof(float) * WIDTH * WIDTH;
    float *mat1 = (float *)malloc(mem_size);
    float *mat2 = (float *)malloc(mem_size);
    float *mat3 = (float *)malloc(mem_size);
    // initialize mat1 and mat2
    fillMat(mat1, WIDTH, WIDTH);
    fillMat(mat2, WIDTH, WIDTH);

    // matrixMul<WIDTH>(mat3, mat1, mat2);

    float* d_mat1, *d_mat2, *d_mat3;
    hipError_t error;
    error = hipMalloc((void**)&d_mat1, mem_size);
    error = hipMemcpy(d_mat1, mat1, mem_size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("hipMemcpy d_mat1 returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(1);
    }
    error = hipMalloc((void**)&d_mat2, mem_size);
    error = hipMemcpy(d_mat2, mat2, mem_size, hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        printf("hipMemcpy d_mat2 returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(1);
    }
    error = hipMalloc((void**)&d_mat3, mem_size);

    int block_size = 32;
    dim3 threads(block_size, block_size);
    dim3 grid(WIDTH/threads.x, WIDTH/threads.y);
    // Allocate CUDA events that we'll use for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matrixMulCUDA<<<grid, threads>>>(d_mat3, d_mat1, d_mat2, WIDTH, WIDTH);

    hipEventRecord(stop);
    float time = 0;
    hipEventElapsedTime(&time, start, stop);
    printf("run time:%.lf ms\n", time);

    error = hipMemcpy(mat3, d_mat3, mem_size, hipMemcpyDeviceToHost);

    print_data(mat1, WIDTH, WIDTH, "mat1");
    print_data(mat2, WIDTH, WIDTH, "mat2");
    print_data(mat3, WIDTH, WIDTH, "mat3");

    free(mat1);
    free(mat2);
    free(mat3);
    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_mat3);
}

int main(int argc, char const *argv[])
{
    // testMatrixMul<256 >();
    // testMatrixMul<512 >();
    // testMatrixMul<768 >();
    testMatrixMul<1024>();
    // testMatrixMul<1280>();
    // testMatrixMul<1536>();
    // testMatrixMul<1792>();
    // testMatrixMul<2048>();
    return 0;
}
