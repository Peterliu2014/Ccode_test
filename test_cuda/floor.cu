
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void tofloor(int n, float* a, float* b, float* smin, float* quant_unit) {
    *b = floor((*a - *smin)/(*quant_unit) + 0.5);
}

__global__
void toclamp(int n, float* a, float* b, float* smin) {
    // *b = clamp(*a, *smin, 0);
    // *b = max(*a, *smin);
    *b = max(0.22199318, -100000000.0);
}

int main() {
    float a = 0.22199318;
    float smin = -100000000;
    float quant_unit = 781250.0;
    float* a_d, *b_d, *smin_d, *quant_unit_d;
    hipMalloc((void**)&a_d, sizeof(float));
    hipMalloc((void**)&smin_d, sizeof(float));
    hipMalloc((void**)&quant_unit_d, sizeof(float));
    hipMalloc((void**)&b_d, sizeof(float));
    hipMemcpy(a_d, &a, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(smin_d, &smin, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(quant_unit_d, &quant_unit, sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(&a, a_d, sizeof(float), hipMemcpyDeviceToHost);
    printf("a:%.lf\n", a);

    int N = 1;
    // tofloor<<<(N+255)/256, 256>>>(N, a_d, b_d, smin_d, quant_unit_d);
    toclamp<<<(N+255)/256, 256>>>(N, a_d, b_d, smin_d);

    float* b;
    b = (float*) malloc(sizeof(float));
    
    hipMemcpy(b, b_d, sizeof(float), hipMemcpyDeviceToHost);
    printf("a:%lf, b:%.lf\n", a, *b);

    free(b);
    hipFree(a_d);
    hipFree(b_d);



    return 0;
}