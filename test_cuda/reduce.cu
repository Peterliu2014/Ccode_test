#include "hip/hip_runtime.h"
#include<cstdio>
#include<vector>
#include<string>
#include<hip/hip_runtime.h>
// #include <thrust/sequence.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h> // thrust::host/device

#define BLOCK_SIZE 256

using namespace std;

float average(const vector<float> &timing) {
  double avg = 0;
  for(vector<float>::const_iterator it = timing.begin(); it != timing.end(); it++) avg += *it;
  avg /= timing.size();

  return avg;
}

void print_info(int *data, int len, string flag) {
    printf("%s frist ten:\n", flag.c_str());
    for (int i=0; i<10; i++){
        printf("%d ", data[i]);
    }
    printf("\n");
    printf("%s last ten:\n", flag.c_str());
    for (int i=len -10; i<len; i++){
        printf("%d ", data[i]);
    }
    printf("\n");
}

void print_device_info(int *device_data, int len, string flag) {
    int*data = (int*)malloc(len * sizeof(int));
    hipMemcpy(data, device_data, len*sizeof(int), hipMemcpyDeviceToHost);

    printf("%s frist ten:\n", flag.c_str());
    for (int i=0; i<10 && i < len; i++){
        printf("%d ", data[i]);
    }
    printf("\n");
    if (len>10) {
        printf("%s last ten:\n", flag.c_str());
        for (int i=len -10; i<len; i++){
            printf("%d ", data[i]);
        }
        printf("\n");
    }
}

// interleved addressing, divergent warps branch
__global__ void reduce_kernel1(int*A, int*out, int len_a) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx>=len_a) return;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ int sA[];
    sA[tid] = A[idx];
    __syncthreads();
    for (int s=1; s<blockDim.x; s*=2) {
        if(tid%(2*s)== 0) {
            if (tid+s < len_a) sA[tid] += sA[tid+s];
        }
        __syncthreads();
    }
    if (tid==0) out[bid] = sA[0];
}

// inerleved addressing, bands conflict
__global__ void reduce_kernel2(int*A, int*out, int len_a) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx>=len_a) return;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ int sA[];
    sA[tid] = A[idx];
    __syncthreads();
    for (int s=1; s<blockDim.x; s*=2) {
        int run_id = 2 * s * tid;
        if(run_id <blockDim.x) {
            if (run_id+s < blockDim.x) sA[run_id] += sA[run_id+s];
        }
        __syncthreads();
    }
    if (tid==0) out[bid] = sA[0];
}

// sequential addressing
__global__ void reduce_kernel3(int*A, int*out, int len_a) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx>=len_a) return;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ int sA[];
    sA[tid] = A[idx];
    __syncthreads();
    for (int s=blockDim.x/2; s>0; s>>=1) {
        if(tid <s) {
            if (tid+s < blockDim.x) sA[tid] += sA[tid+s];
        }
        __syncthreads();
    }
    if (tid==0) out[bid] = sA[0];
}

// first add during load
__global__ void reduce_kernel4(int*A, int*out, int len_a) {
    int idx = blockDim.x * (blockIdx.x * 2) + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ int sA[];
    if (idx + blockDim.x < len_a) {
        sA[tid] = A[idx] + A[blockDim.x + idx];
    }
    else{
        sA[tid] = A[idx];
    }
    
    __syncthreads();
    for (int s=blockDim.x/2; s>0; s>>=1) {
        if(tid <s) {
            if (tid+s < blockDim.x) sA[tid] += sA[tid+s];
        }
        __syncthreads();
    }
    if (tid==0) out[bid] = sA[0];
}

__global__ void copy(int*A, int*out, int len_a) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= len_a) return;
    out[idx] = A[idx];
}

int reduce(int*A, int len_a, int numThreads){
    // numBlocks = (len_a + numThreads) / numThreads;
    int numBlocks = (len_a + numThreads*2 -1) / numThreads/2;
    int*tmp;
    hipMalloc((void**)&tmp, numBlocks*sizeof(int));
    while(len_a>1) {
        printf("len:%d, numBlocks:%d, numThreads:%d\n", len_a, numBlocks, numThreads);
        // reduce_kernel1<<<numBlocks, numThreads, numThreads*sizeof(int)>>>(A, tmp, len_a);
        // reduce_kernel2<<<numBlocks, numThreads, numThreads*sizeof(int)>>>(A, tmp, len_a);
        // reduce_kernel3<<<numBlocks, numThreads, numThreads*sizeof(int)>>>(A, tmp, len_a);
        // len_a = numBlocks;
        // numBlocks = (numBlocks+numThreads-1)/numThreads;

        reduce_kernel4<<<numBlocks, numThreads, numThreads*sizeof(int)>>>(A, tmp, len_a);
        len_a = numBlocks;
        numBlocks = (len_a + numThreads*2 -1) / numThreads/2;
        copy<<<numBlocks, numThreads>>>(tmp, A, len_a);
        // print_device_info(tmp, len_a, "tmp");
    }
    int result;
    hipMemcpy(&result, tmp, sizeof(int), hipMemcpyDeviceToHost);
    return result;
}


int main() {
    // int len_a = 256 * 256 * 256;
    int len_a = 1024*1024*1024;
    int numThreads = BLOCK_SIZE;
    int numBlocks = (len_a + numThreads - 1) / numThreads;

    int* A = (int*) malloc(len_a * sizeof(int));
    int* th_h_A = (int*) malloc(len_a * sizeof(int));
    // file a,b
    for (int i=0; i<len_a; i++) A[i] = 1;
    for (int i=0; i<len_a; i++) th_h_A[i] = 1;
    // print_info(A, len_a, "A");

    int *d_A, *th_A;
    hipMalloc((void**)&d_A, len_a * sizeof(int));
    hipMalloc((void**)&th_A, len_a * sizeof(int));

    hipMemcpy(d_A, A, len_a * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(th_A, A, len_a * sizeof(int), hipMemcpyHostToDevice);

    int th_result;
    // thrust reduce
    th_result = thrust::reduce(thrust::device, th_A, th_A+len_a, 0);
    // th_result = thrust::reduce(thrust::host, th_h_A, th_h_A+len_a, 0);
    printf("th result:%d\n", th_result);

    vector<float> times;
    int loops = 1;
    for (int i=0; i<loops; i++) {
        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start, NULL);

        int result = reduce(d_A, len_a, numThreads);
        printf("result:%d\n", result);

        hipEventRecord(end, NULL);
        hipEventSynchronize(end);
        float time = 0;
        hipEventElapsedTime(&time, start, end);
        times.push_back(time);
    }
    printf("reduce avg time:%lf\n", average(times));

}