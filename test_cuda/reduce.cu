#include "hip/hip_runtime.h"
#include<cstdio>
#include<vector>
#include<string>
#include<hip/hip_runtime.h>
// #include <thrust/sequence.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h> // thrust::host/device

#define BLOCK_SIZE 256

using namespace std;

float average(const vector<float> &timing) {
  double avg = 0;
  for(vector<float>::const_iterator it = timing.begin(); it != timing.end(); it++) avg += *it;
  avg /= timing.size();

  return avg;
}

void print_info(int *data, int len, string flag) {
    printf("%s frist ten:\n", flag.c_str());
    for (int i=0; i<10; i++){
        printf("%d ", data[i]);
    }
    printf("\n");
    printf("%s last ten:\n", flag.c_str());
    for (int i=len -10; i<len; i++){
        printf("%d ", data[i]);
    }
    printf("\n");
}

void print_device_info(int *device_data, int len, string flag) {
    int*data = (int*)malloc(len * sizeof(int));
    hipMemcpy(data, device_data, len*sizeof(int), hipMemcpyDeviceToHost);

    printf("%s frist ten:\n", flag.c_str());
    for (int i=0; i<10 && i < len; i++){
        printf("%d ", data[i]);
    }
    printf("\n");
    if (len>10) {
        printf("%s last ten:\n", flag.c_str());
        for (int i=len -10; i<len; i++){
            printf("%d ", data[i]);
        }
        printf("\n");
    }
}

// interleved addressing, divergent warps branch
// time: 33.3 ms
__global__ void reduce_kernel1(int*A, int*out, int len_a) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ int sA[];
    sA[tid] = 0;
    if (idx < len_a) sA[tid] = A[idx];
    __syncthreads();
    for (int s=1; s<blockDim.x; s*=2) {
        if(tid%(2*s)== 0) {
            if (tid+s < len_a) sA[tid] += sA[tid+s];
        }
        __syncthreads();
    }
    if (tid==0) out[bid] = sA[0];
}

// inerleved addressing, bands conflict
// time: 20 ms
__global__ void reduce_kernel2(int*A, int*out, int len_a) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ int sA[];
    sA[tid] = 0;
    if (idx < len_a) sA[tid] = A[idx];
    __syncthreads();
    for (int s=1; s<blockDim.x; s*=2) {
        int run_id = 2 * s * tid;
        if(run_id <blockDim.x) {
            if (run_id+s < blockDim.x) sA[run_id] += sA[run_id+s];
        }
        __syncthreads();
    }
    if (tid==0) out[bid] = sA[0];
}

// sequential addressing
// time: 17.9 ms
__global__ void reduce_kernel3(int*A, int*out, int len_a) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ int sA[];
    sA[tid] = 0;
    if (idx < len_a) sA[tid] = A[idx];
    __syncthreads();
    for (int s=blockDim.x/2; s>0; s>>=1) {
        if(tid <s) {
          sA[tid] += sA[tid+s];
        }
        __syncthreads();
    }
    if (tid==0) out[bid] = sA[0];
}

// first add during load
// time:10.78 ms
__global__ void reduce_kernel4(int*A, int*out, int len_a) {
    int idx = blockDim.x * (blockIdx.x * 2) + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ int sA[];
    sA[tid] = 0;
    if (idx < len_a) sA[tid] = A[idx];
    if (idx + blockDim.x < len_a) sA[tid] += A[idx + blockDim.x];
    
    __syncthreads();
    for (int s=blockDim.x/2; s>0; s>>=1) {
        if(tid <s) {
            sA[tid] += sA[tid+s];
        }
        __syncthreads();
    }
    if (tid==0) out[bid] = sA[0];
}

// unroll the last warp
__device__ void warpFunc(volatile int* sdata, int tid) {
    sdata[tid] += sdata[tid+32];
    sdata[tid] += sdata[tid+16];
    sdata[tid] += sdata[tid+8];
    sdata[tid] += sdata[tid+4];
    sdata[tid] += sdata[tid+2];
    sdata[tid] += sdata[tid+1];
}
// time: 8.0 ms
__global__ void reduce_kernel5(int *A, int *out, int len_a) {
    int idx = blockDim.x * (blockIdx.x * 2) + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ int sA[];
    sA[tid] = 0;
    if (idx < len_a) sA[tid] = A[idx];
    if (idx + blockDim.x < len_a) sA[tid] += A[idx + blockDim.x];
    
    __syncthreads();
    for (int s=blockDim.x/2; s>32; s>>=1) {
        if(tid <s) {
            sA[tid] += sA[tid+s];
        }
        __syncthreads();
    }
    if (tid<32) warpFunc(sA, tid);
    if (tid==0) out[bid] = sA[0];
}

// completely unroll
// time: 8.03 ms
__global__ void reduce_kernel6(int *A, int *out, int len_a) {
    int idx = blockDim.x * (blockIdx.x * 2) + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ int sA[];
    sA[tid] = 0;
    if (idx < len_a) sA[tid] = A[idx];
    if (idx + blockDim.x < len_a) sA[tid] += A[idx + blockDim.x];
    __syncthreads();

    if (blockDim.x >= 1024) { if (tid < 512) sA[tid] += sA[tid+512]; __syncthreads();}
    if (blockDim.x >= 512) { if (tid < 256) sA[tid] += sA[tid+256]; __syncthreads();}
    if (blockDim.x >= 256) { if (tid < 128) sA[tid] += sA[tid+128]; __syncthreads();}
    if (blockDim.x >= 128) { if (tid < 64) sA[tid] += sA[tid+64]; __syncthreads();}
    if (tid<32) warpFunc(sA, tid);
    // if (tid<32) {
    //     sA[tid]+= sA[tid+32];
    //     sA[tid]+= sA[tid+16];
    //     sA[tid]+= sA[tid+8];
    //     sA[tid]+= sA[tid+4];
    //     sA[tid]+= sA[tid+2];
    //     sA[tid]+= sA[tid+1];
    // }
    if (tid==0) out[bid] = sA[0];
}

template <unsigned int blockSize>
__device__ void warpReduce(volatile int *sdata, int tid) {
    if (blockSize >=64) sdata[tid] += sdata[tid+32];
    if (blockSize >=32) sdata[tid] += sdata[tid+16];
    if (blockSize >=16) sdata[tid] += sdata[tid+8];
    if (blockSize >=8) sdata[tid] += sdata[tid+4];
    if (blockSize >=4) sdata[tid] += sdata[tid+2];
    if (blockSize >=2) sdata[tid] += sdata[tid+1];
}

// load as much as data
__global__ void reduce_kernel7(int *A, int *out, int len_a) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int griz = gridDim.x * blockDim.x;
    extern __shared__ int sA[];
    sA[tid] = 0;
    int tmp_id = idx;
    while(tmp_id <len_a) {
        if (tmp_id < len_a) sA[tid] = A[tmp_id];
        tmp_id += griz;
    }
    __syncthreads();
    if (blockDim.x >= 1024) { if (tid < 512) sA[tid] += sA[tid+512]; __syncthreads();}
    if (blockDim.x >= 512) { if (tid < 256) sA[tid] += sA[tid+256]; __syncthreads();}
    if (blockDim.x >= 256) { if (tid < 128) sA[tid] += sA[tid+128]; __syncthreads();}
    if (blockDim.x >= 128) { if (tid < 64) sA[tid] += sA[tid+64]; __syncthreads();}
    if (tid<32) warpFunc(sA, tid);
    if (tid==0) out[bid] = sA[0];
}


int reduce(int*A, int len_a, int numThreads){
    int numBlocks = (len_a + numThreads) / numThreads;
    // int numBlocks = (len_a + numThreads*2 -1) / numThreads/2;
    int*tmp;
    hipMalloc((void**)&tmp, numBlocks*sizeof(int));
    while(len_a>1) {
        printf("len:%d, numBlocks:%d, numThreads:%d\n", len_a, numBlocks, numThreads);
        // reduce_kernel1<<<numBlocks, numThreads, numThreads*sizeof(int)>>>(A, tmp, len_a);
        // // time: 33.3 ms

        // reduce_kernel2<<<numBlocks, numThreads, numThreads*sizeof(int)>>>(A, tmp, len_a);
        // // time: 20 ms

        reduce_kernel3<<<numBlocks, numThreads, numThreads*sizeof(int)>>>(A, tmp, len_a);
        // time: 17.9 ms
        
        len_a = numBlocks;
        numBlocks = (numBlocks+numThreads-1)/numThreads;
        hipMemcpy(A, tmp, sizeof(int) * len_a, hipMemcpyDeviceToDevice);
        // print_device_info(tmp, len_a, "tmp");
    }
    int result;
    hipMemcpy(&result, tmp, sizeof(int), hipMemcpyDeviceToHost);
    return result;
}

int reduce_2(int*A, int len_a, int numThreads){
    int numBlocks = (len_a + numThreads*2 -1) / numThreads/2;
    int*tmp;
    hipMalloc((void**)&tmp, numBlocks*sizeof(int));
    while(len_a>1) {
        printf("len:%d, numBlocks:%d, numThreads:%d\n", len_a, numBlocks, numThreads);
        
        // reduce_kernel4<<<numBlocks, numThreads, numThreads*sizeof(int)>>>(A, tmp, len_a);
        // // time:10.78 ms

        // reduce_kernel5<<<numBlocks, numThreads, numThreads*sizeof(int)>>>(A, tmp, len_a);
        // // time: 8.0 ms

        reduce_kernel6<<<numBlocks, numThreads, numThreads*sizeof(int)>>>(A, tmp, len_a);
        // time: 8.03 ms

        len_a = numBlocks;
        numBlocks = (numBlocks+numThreads*2-1)/numThreads/2;
        hipMemcpy(A, tmp, sizeof(int) * len_a, hipMemcpyDeviceToDevice);
        // print_device_info(tmp, len_a, "tmp");
    }
    int result;
    hipMemcpy(&result, tmp, sizeof(int), hipMemcpyDeviceToHost);
    return result;
}

int reduce_3(int*A, int len_a, int numThreads){
    // int count_per_threads = 2;
    // int count_per_blocks = 2;
    int numBlocks = (len_a + numThreads*8 -1) / numThreads/8;
    int*tmp;
    hipMalloc((void**)&tmp, numBlocks*sizeof(int));
    while(len_a>1) {
        printf("len:%d, numBlocks:%d, numThreads:%d\n", len_a, numBlocks, numThreads);
        reduce_kernel7<<<numBlocks, numThreads, numThreads*sizeof(int)>>>(A, tmp, len_a);
        // time:  ms

        len_a = numBlocks;
        numBlocks = (numBlocks+numThreads*8-1)/numThreads/8;
        hipMemcpy(A, tmp, sizeof(int) * len_a, hipMemcpyDeviceToDevice);
        // print_device_info(tmp, len_a, "tmp");
    }
    int result;
    hipMemcpy(&result, tmp, sizeof(int), hipMemcpyDeviceToHost);
    return result;
}


int main() {
    // int len_a = 256 * 256 * 256;
    int len_a = 1024*1024*1024;
    int numThreads = BLOCK_SIZE;

    int* A = (int*) malloc(len_a * sizeof(int));
    int* th_h_A = (int*) malloc(len_a * sizeof(int));
    // file a,b
    for (int i=0; i<len_a; i++) A[i] = 1;
    for (int i=0; i<len_a; i++) th_h_A[i] = 1;
    // print_info(A, len_a, "A");

    int *d_A, *th_A;
    hipMalloc((void**)&d_A, len_a * sizeof(int));
    hipMalloc((void**)&th_A, len_a * sizeof(int));

    hipMemcpy(d_A, A, len_a * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(th_A, A, len_a * sizeof(int), hipMemcpyHostToDevice);

    int th_result;
    // thrust reduce
    th_result = thrust::reduce(thrust::device, th_A, th_A+len_a, 0);
    // th_result = thrust::reduce(thrust::host, th_h_A, th_h_A+len_a, 0);
    printf("th result:%d\n", th_result);

    vector<float> times;
    int loops = 1;
    for (int i=0; i<loops; i++) {
        hipEvent_t start, end;
        hipEventCreate(&start);
        hipEventCreate(&end);
        hipEventRecord(start, NULL);

        // int result = reduce(d_A, len_a, numThreads);
        int result = reduce_2(d_A, len_a, numThreads);
        printf("result:%d\n", result);

        hipEventRecord(end, NULL);
        hipEventSynchronize(end);
        float time = 0;
        hipEventElapsedTime(&time, start, end);
        times.push_back(time);
    }
    printf("reduce avg time:%lf\n", average(times));

    free(A);
    free(th_h_A);
    hipFree(d_A);
    hipFree(th_A);

}