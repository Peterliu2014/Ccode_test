
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void revert(int n, float* a, float *b) {
    *b = - (*a);
}

int main() {
    float* a, *b, *a_d, *b_d;
    a = (float*)malloc(sizeof(float));
    b = (float*)malloc(sizeof(float));
    *a = 5.0f;
    *b = 0;
    hipMalloc((void**)&a_d, sizeof(float));
    hipMalloc((void**)&b_d, sizeof(float));
    hipMemcpy(a_d, a, sizeof(float), hipMemcpyHostToDevice);
    // *b_d = *a_d;
    int N = 1;
    revert<<<(N+255)/256, 256>>>(N, a_d, b_d);
    hipMemcpy(b, b_d, sizeof(float), hipMemcpyDeviceToHost);

    // *a = -(*a);
    // cudaMemcpy(b_d, a, sizeof(float), cudaMemcpyHostToDevice);
    // cudaMemcpy(b, b_d, sizeof(float), cudaMemcpyDeviceToHost);

    
    printf("a:%.2f, b:%.2f\n", *a, *b);
    return 0;
}


// __global__
// void saxpy(int n, float a, float *x, float *y)
// {
//   int i = blockIdx.x*blockDim.x + threadIdx.x;
//   if (i < n) y[i] = a*x[i] + y[i];
// }

// int main(void)
// {
//   int N = 1<<20;
//   float *x, *y, *d_x, *d_y;
//   x = (float*)malloc(N*sizeof(float));
//   y = (float*)malloc(N*sizeof(float));

//   cudaMalloc(&d_x, N*sizeof(float)); 
//   cudaMalloc(&d_y, N*sizeof(float));

//   for (int i = 0; i < N; i++) {
//     x[i] = 1.0f;
//     y[i] = 2.0f;
//   }

//   cudaMemcpy(d_x, x, N*sizeof(float), cudaMemcpyHostToDevice);
//   cudaMemcpy(d_y, y, N*sizeof(float), cudaMemcpyHostToDevice);

//   // Perform SAXPY on 1M elements
//   saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

//   cudaMemcpy(y, d_y, N*sizeof(float), cudaMemcpyDeviceToHost);

//   float maxError = 0.0f;
//   for (int i = 0; i < N; i++)
//     maxError = max(maxError, abs(y[i]-4.0f));
//   printf("Max error: %f\n", maxError);

//   cudaFree(d_x);
//   cudaFree(d_y);
//   free(x);
//   free(y);
// }